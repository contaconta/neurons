#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */
 
 

#include <assert.h>
#include <cutil_inline.h>
// #include <vector>


////////////////////////////////////////////////////////////////////////////////
// Convolution kernel storage
////////////////////////////////////////////////////////////////////////////////
__constant__ float c_Kernel_h[100];
__constant__ float c_Kernel_v[100];
__constant__ float c_Kernel_d[100];

extern "C" void setConvolutionKernel_horizontal(float *h_Kernel, int kernel_length){
  hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel_h), h_Kernel, kernel_length * sizeof(float));
}
extern "C" void setConvolutionKernel_vertical(float *h_Kernel, int kernel_length){
  hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel_v), h_Kernel, kernel_length * sizeof(float));
}

extern "C" void setConvolutionKernel_depth(float *h_Kernel, int kernel_length){
  hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel_d), h_Kernel, kernel_length * sizeof(float));
}

////////////////////////////////////////////////////////////////////////////////
// Constants
////////////////////////////////////////////////////////////////////////////////
#define   ROWS_BLOCKDIM_X 16
#define   ROWS_BLOCKDIM_Y 16
#define   ROWS_RESULT_STEPS 8
#define   ROWS_HALO_STEPS 3
#define   COLUMNS_BLOCKDIM_X 16
#define   COLUMNS_BLOCKDIM_Y 16
#define   COLUMNS_RESULT_STEPS 8
#define   COLUMNS_HALO_STEPS 3
#define   DEPTH_BLOCKDIM_Y 16
#define   DEPTH_BLOCKDIM_Z 16
#define   DEPTH_RESULT_STEPS 4
#define   DEPTH_HALO_STEPS 3



////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int imageD,
    int kernel_radius
){
    __shared__ float s_Data[ROWS_BLOCKDIM_Y]
                           [(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    //Offset to the left halo edge
    int n_blocks_per_row = imageW/(ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X);
    int basez = floor(float(blockIdx.x)/n_blocks_per_row);

    int blockx = blockIdx.x - basez*n_blocks_per_row;
    // const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) *
                      // ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseX = (blockx * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) *
                      ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

    d_Src += basez*imageW*imageH + baseY * imageW + baseX;
    d_Dst += basez*imageW*imageH + baseY * imageW + baseX;

    //Main data
    #pragma unroll
    for(int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
        s_Data[threadIdx.y]
              [threadIdx.x + i * ROWS_BLOCKDIM_X]
          = d_Src[i * ROWS_BLOCKDIM_X];

    //Left halo
    for(int i = 0; i < ROWS_HALO_STEPS; i++){
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] =
            (baseX >= -i * ROWS_BLOCKDIM_X ) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Right halo
    for(int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS;
        i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++){
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] =
            (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Compute and store results
    __syncthreads();
    // #pragma unroll
    for(int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++){
        float sum = 0;

        #pragma unroll
        for(int j = -kernel_radius; j <= kernel_radius; j++)
            sum += c_Kernel_h[kernel_radius - j] * 
                   s_Data    [threadIdx.y]
                             [threadIdx.x + i * ROWS_BLOCKDIM_X + j];
        d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}

extern "C" void convolutionRowsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int imageD,
    int kernel_radius
){
    assert( ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= kernel_radius );
    //There is a rational division of the image into blocks
    assert( imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0 );
    assert( imageH % ROWS_BLOCKDIM_Y == 0 );

    dim3 blocks(imageD*(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X)),
                imageH / ROWS_BLOCKDIM_Y);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);
    convolutionRowsKernel<<<blocks, threads>>>(
                                               // &d_Dst[i*imageH*imageW],
                                               // &d_Src[i*imageH*imageW],
                                               d_Dst,
                                               d_Src,
                                               imageW,
                                               imageH,
                                               imageD,
                                               kernel_radius
                                               );
    cutilCheckMsg("convolutionRowsKernel() execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch,
    int kernel_radius
){
    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

    int n_blocks_per_column = imageH/(COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y);
    int basez = floor(float(blockIdx.y)/n_blocks_per_column);
    int blocky = blockIdx.y - basez*n_blocks_per_column;

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blocky * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    d_Src += basez*imageH*imageW + baseY * imageH + baseX;
    d_Dst += basez*imageH*imageW + baseY * imageH + baseX;

    //Main data
    #pragma unroll
    for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * pitch];

    //Upper halo
    for(int i = 0; i < COLUMNS_HALO_STEPS; i++)
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = 
            (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;

    //Lower halo
    for(int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] =
            (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;

    //Compute and store results
    __syncthreads();
    // #pragma unroll
    for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++){
        float sum = 0;
        #pragma unroll
        for(int j = -kernel_radius; j <= kernel_radius; j++)
            sum += c_Kernel_v[kernel_radius - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];

        d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
    }
}

extern "C" void convolutionColumnsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int imageD,
    int kernel_radius
){
    assert( COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= kernel_radius );
    assert( imageW % COLUMNS_BLOCKDIM_X == 0 );
    assert( imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0 );

    dim3 blocks(imageW / COLUMNS_BLOCKDIM_X, imageD * imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);

    convolutionColumnsKernel<<<blocks, threads>>>(
                                                  d_Dst,
                                                  d_Src,
                                                  imageW,
                                                  imageH,
                                                  imageW,
                                                  kernel_radius
                                                  );
    cutilCheckMsg("convolutionColumnsKernel() execution failed\n");
}

////////////////////////////////////////////////////////////////////////////////
// Depth convolution filter - Really naive implementation
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionDepthKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int imageD,
    int kernel_radius
){
    __shared__ float s_Data[DEPTH_BLOCKDIM_Y]
                           [(DEPTH_RESULT_STEPS + 2 * DEPTH_HALO_STEPS) * DEPTH_BLOCKDIM_Z];

    //Offset to the left halo edge
    int n_blocks_per_depth = imageD / (DEPTH_RESULT_STEPS * DEPTH_BLOCKDIM_Z);
    int basex = floor(float(blockIdx.x)/n_blocks_per_depth);

    int blockz = blockIdx.x - basex*n_blocks_per_depth;

    const int baseZ = (blockz * DEPTH_RESULT_STEPS - DEPTH_HALO_STEPS)*DEPTH_BLOCKDIM_Z +
                      threadIdx.x;
    const int baseY = blockIdx.y * DEPTH_BLOCKDIM_Y + threadIdx.y;

    //Put the pointers to the beginning of the data
    d_Src += baseZ * imageW * imageH + baseY * imageW + basex;
    d_Dst += baseZ * imageW * imageH + baseY * imageW + basex;

    // //Main data
    #pragma unroll
    for(int i = DEPTH_HALO_STEPS; i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS; i++)
        s_Data[threadIdx.y]
              [threadIdx.x + i * DEPTH_BLOCKDIM_Z]
          = d_Src[i * DEPTH_BLOCKDIM_Z * imageH * imageW];

    //Left halo
    for(int i = 0; i < DEPTH_HALO_STEPS; i++){
        s_Data[threadIdx.y][threadIdx.x + i * DEPTH_BLOCKDIM_Z] =
            (baseZ >= -i * DEPTH_BLOCKDIM_Z ) ? 
          d_Src[i * DEPTH_BLOCKDIM_Z * imageH * imageW] : 0;
    }

    // Right halo
    for(int i = DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS;
        i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS + DEPTH_HALO_STEPS; i++){
        s_Data[threadIdx.y][threadIdx.x + i * DEPTH_BLOCKDIM_Z] =
            (imageD - baseZ > i * DEPTH_BLOCKDIM_Z ) ? 
          d_Src[i * DEPTH_BLOCKDIM_Z * imageH * imageW] : 0;
    }

    // //Compute and store results
    __syncthreads();
    #pragma unroll
    for(int i = DEPTH_HALO_STEPS; i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS; i++){
        float sum = 0;
        #pragma unroll
        for(int j = -kernel_radius; j <= kernel_radius; j++)
            sum += c_Kernel_d[kernel_radius - j] * 
                   s_Data    [threadIdx.y]
                             [threadIdx.x + i * DEPTH_BLOCKDIM_Z + j];
        d_Dst[i * DEPTH_BLOCKDIM_Z * imageH * imageW] = sum;
    }
}

extern "C" void convolutionDepthGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int imageD,
    int kernel_radius
){
    assert( DEPTH_BLOCKDIM_Z * DEPTH_HALO_STEPS >= kernel_radius );
    //There is a rational division of the image into blocks
    assert( imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0 );
    assert( imageH % ROWS_BLOCKDIM_Y == 0 );

    dim3 blocks(imageW*imageD / (DEPTH_RESULT_STEPS * DEPTH_BLOCKDIM_Z),
                imageH / DEPTH_BLOCKDIM_Y);
    dim3 threads(DEPTH_BLOCKDIM_Z, DEPTH_BLOCKDIM_Y);

    // for(int x = 0; x < imageW; x++)
      convolutionDepthKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageD,
        kernel_radius
    );
    cutilCheckMsg("convolutionRowsKernel() execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Computes the higher eigenvalue of the hessian
////////////////////////////////////////////////////////////////////////////////
__global__ void hessianKernel
(
 float *d_output,
 float *d_gxx,
 float *d_gxy,
 float *d_gxz,
 float *d_gyy,
 float *d_gyz,
 float *d_gzz,
 int imageW,
 int imageH,
 int imageD,
){
  int z = ceil(blockIdx.x/ROWS_BLOCKDIM_X);
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  int x = (blockIdx.x - z*ROWS_BLOCKDIM_X)*blockDim.x + threadIdx.x;
  int i = z*imageW*imageH + y*imageW + x
  float a, b, c;
  a = d_gxx[i];
  b = d_gxy[i];
  c = d_gyy[i];
  d_output[i] = (a+c)/2 + sqrt( (a-c)*(a-c) + 4*b*b)/2;
  // d_output[i] = b;
}



extern "C" void hessianGPU
(
 float *d_output,
 float *d_gxx,
 float *d_gxy,
 float *d_gxz,
 float *d_gyy,
 float *d_gyz,
 float *d_gzz,
 int imageW,
 int imageH,
 int imageD,
 )
{
  dim3 gird (imageD*ceil(float(imageW)/ROWS_BLOCKDIM_X),ceil(float(imageH)/ROWS_BLOCKDIM_Y));
  dim3 block(ROWS_BLOCKDIM_X,ROWS_BLOCKDIM_Y);
  hessianKernel<<<gird, block>>>( d_output, d_gxx, d_gxy, d_gxz,
                                  d_gyy, d_gyz, d_gzz, imageW, imageH, imageD );
  cutilCheckMsg("hessianKernel() execution failed\n");
}
